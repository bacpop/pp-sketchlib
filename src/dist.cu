#include "hip/hip_runtime.h"
/*
 *
 * dist.cpp
 * PopPUNK dists using CUDA
 *
 */

// std
#include <cstdint>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <cmath>
#include <stdexcept>
#include <vector>
#include <tuple>
#include <algorithm>
#include <iomanip>
#include <chrono>
#include <ctime>
#include <ratio>

// cuda
#include <thrust/device_vector.h>
#include <thrust/copy.h>

// internal headers
#include "bitfuncs.hpp"
#include "gpu.hpp"

const int WARP_SIZE = 32;
const int selfBlockSize = 32;
const float mem_epsilon = 0.05;

struct DeviceMemory {
	thrust::device_vector<uint64_t> ref_sketches;	
	thrust::device_vector<uint64_t> query_sketches;	
	thrust::device_vector<float> ref_random;	
	thrust::device_vector<float> query_random;	
	thrust::device_vector<int> kmers;	
	thrust::device_vector<float> dist_mat;	
};

// Structure of flattened vectors
struct SketchStrides {
	size_t bin_stride;
	size_t kmer_stride;
	size_t sample_stride;
	size_t sketchsize64; 
	size_t bbits;
};

/******************
*			      *
*	Device code   *
*			      *	
*******************/

// Error checking of dynamic memory allocation on device
// https://stackoverflow.com/a/14038590
#define cdpErrchk(ans) { cdpAssert((ans), __FILE__, __LINE__); }
__device__ void cdpAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) assert(0);
   }
}

// Ternary used in observed_excess
template <class T>
__device__
T non_neg_minus(T a, T b) {
	return a > b ? (a - b) : 0;
}

// Calculates excess observations above a random value
template <class T>
__device__
T observed_excess(T obs, T exp, T max) {
	T diff = non_neg_minus(obs, exp);
	return(diff * max / (max - exp));
}

// CUDA version of bindash dist function (see dist.cpp)
__device__
float jaccard_dist(const uint64_t * sketch1, 
                   const uint64_t * sketch2, 
				   const SketchStrides& s1_strides, 
				   const SketchStrides& s2_strides) 
{
	size_t samebits = 0;
    for (int i = 0; i < s1_strides.sketchsize64; i++) 
    {
		uint64_t bits = ~((uint64_t)0ULL);
		for (int j = 0; j < s1_strides.bbits; j++) 
        {
			long long bin_index = i * s1_strides.bbits + j;
			bits &= ~(sketch1[bin_index * s1_strides.bin_stride] ^ sketch2[bin_index * s2_strides.bin_stride]);
		}

		samebits += __popcll(bits); // CUDA 64-bit popcnt
	}
	const size_t maxnbits = s1_strides.sketchsize64 * NBITS(uint64_t); 
	const size_t expected_samebits = (maxnbits >> s1_strides.bbits);
	size_t intersize = samebits;
	if (!expected_samebits) 
	{
		size_t ret = observed_excess(samebits, expected_samebits, maxnbits);
	}
	size_t unionsize = NBITS(uint64_t) * s1_strides.sketchsize64;
    float jaccard = __fdiv_ru(intersize, unionsize);
    return(jaccard);
}

// Simple linear regression, exact solution
// Avoids use of dynamic memory allocation on device, or
// linear algebra libraries
__device__
void simple_linear_regression(float * const &core_dist,
				              float * const &accessory_dist,
							  const float xsum,
							  const float ysum,
							  const float xysum,
							  const float xsquaresum,
							  const float ysquaresum,
							  const int n)
{
	// Here I use CUDA fast-math intrinsics on floats, which give comparable accuracy
	// --use-fast-math compile option also possible, but gives less control
	// __fmul_ru(x, y) = x * y and rounds up. 
	// __fpow(x, a) = x^a give 0 for x<0, so not using here (and it is slow)
	// could also replace add / subtract, but becomes less readable
	float xbar = xsum / n;
	float ybar = ysum / n;
    float x_diff = xsquaresum - __fmul_ru(xsum, xsum)/n;
    float y_diff = ysquaresum - __fmul_ru(ysum, ysum)/n;
	float xstddev = __fsqrt_ru((xsquaresum - __fmul_ru(xsum, xsum)/n)/n);
	float ystddev = __fsqrt_ru((ysquaresum - __fmul_ru(ysum, ysum)/n)/n);
	float r = __fdiv_ru(xysum - __fmul_ru(xsum, ysum)/n,  __fsqrt_ru(x_diff*y_diff));
	float beta = __fmul_ru(r, __fdiv_ru(ystddev, xstddev));
    float alpha = __fmaf_ru(-beta, xbar, ybar); // maf: x * y + z

	// Store core/accessory in dists, truncating at zero
	if (beta < 0) {
		*core_dist = 1 - __expf(beta);
	} else {
		*core_dist = 0;
	}

	if (alpha < 0) {
		*accessory_dist = 1 - __expf(alpha);
	} else {
		*accessory_dist = 0;
	}
}

// Functions to convert index position to/from squareform to condensed form
__device__
long calc_row_idx(const long long k, const long n) {
	// __ll2float_rn() casts long long to float, rounding to nearest
	return n - 2 - floor(__dsqrt_rn(__ll2double_rz(-8*k + 4*n*(n-1)-7))/2 - 0.5);
}

__device__
long calc_col_idx(const long long k, const long i, const long n) {
	return k + i + 1 - n*(n-1)/2 + (n-i)*((n-i)-1)/2;
}

__device__
long long square_to_condensed(long i, long j, long n) {
    assert(i > j);
	return (n*j - ((j*(j+1)) >> 1) + i - 1 - j);
}

/******************
*			      *
*	Global code   *
*			      *	
*******************/

// Main kernel functions run on the device, 
// but callable from the host

// To calculate distance of query sketches from a panel
// of references
__global__
void calculate_query_dists(const uint64_t * ref,
					 const long ref_n,
					 const uint64_t * query,
					 const long query_n,
					 const int * kmers,
					 const int kmer_n,
					 float * dists,
					 const long long dist_n,
					 const float * random_match_ref,
					 const float * random_match_query,
					 const SketchStrides ref_strides,
					 const SketchStrides query_strides) {
	// Calculate indices for query, ref and results
	long blocksPerQuery = (ref_n + blockDim.x - 1) / blockDim.x;
	long query_idx = __float2int_rz(__fdividef(blockIdx.x, blocksPerQuery) + 0.001f);
	long ref_idx = (blockIdx.x % blocksPerQuery) * blockDim.x + threadIdx.x;
	long dist_idx = query_idx * ref_n + ref_idx;
	const uint64_t* ref_start = ref + ref_idx * ref_strides.sample_stride;
	const uint64_t* query_start = query + query_idx * query_strides.sample_stride;
	
	// Calculate Jaccard distances over k-mer lengths
	float xsum = 0; float ysum = 0; float xysum = 0;
	float xsquaresum = 0; float ysquaresum = 0;
	for (int kmer_idx = 0; kmer_idx < kmer_n; kmer_idx++)
	{
		// Copy query sketch into __shared__ mem (on chip) for faster access within block
		// Hopefully this doesn't suffer from bank conflicts as the sketch2 access in
		// jaccard_distance() should result in a broadcast
		// Uses all threads *in a single warp* to do the copy
		// NB there is no disadvantage vs using multiple warps, as they would have to wait
		// (see https://stackoverflow.com/questions/15468059/copy-to-the-shared-memory-in-cuda)
		extern __shared__ uint64_t query_shared[];
		if (threadIdx.x < WARP_SIZE) {
			for (long lidx = threadIdx.x; lidx < query_strides.bbits * query_strides.sketchsize64; lidx += WARP_SIZE) {
				query_shared[lidx] = query_start[lidx * query_strides.bin_stride];
			}
		}
		__syncthreads();
	
		// Some threads at the end of the last block will have nothing to do
		// Need to have conditional here to avoid block on __syncthreads() above
		if (ref_idx < ref_n)
		{
			// Calculate Jaccard distance at current k-mer length
			float jaccard_obs = jaccard_dist(ref_start, query_start, ref_strides, query_strides);

			// Adjust for random matches
			float r1 = random_match_ref[kmer_idx * ref_n + ref_idx];
			float r2 = random_match_query[kmer_idx * query_n + query_idx];
			float jaccard_expected = (r1 * r2) / (r1 + r2 - r1 * r2);
			float y = __logf(observed_excess(jaccard_obs, jaccard_expected, 1.0f));

			// Running totals for regression
			xsum += kmers[kmer_idx]; 
			ysum += y; 
			xysum += kmers[kmer_idx] * y;
			xsquaresum += kmers[kmer_idx] * kmers[kmer_idx];
			ysquaresum += y * y;
		}

		// Move to next k-mer length
		ref_start += ref_strides.kmer_stride;
		query_start += query_strides.kmer_stride;
	}

	if (ref_idx < ref_n)
	{
		// Run the regression, and store results in dists
		simple_linear_regression(dists + dist_idx,
								 dists + dist_n + dist_idx,
								 xsum,
								 ysum,
								 xysum,
								 xsquaresum,
								 ysquaresum,
								 kmer_n);

		// Progress indicator
		// The >> 10 is a divide by 1024 - update roughly every 0.1%
		if (dist_idx % (dist_n >> 10) == 0) 
		{
			printf("%cProgress (GPU): %.1lf%%", 13, (float)dist_idx/dist_n * 100);
		}
	}

}

// Takes a position in the condensed form distance matrix, converts into an
// i, j for the ref/query vectors. Calls regression with these start points
__global__
void calculate_self_dists(const uint64_t * ref,
					      const long ref_n,
					      const int * kmers,
					      const int kmer_n,
					      float * dists,
						  const long long dist_n,
						  const float * random_match,
					      const SketchStrides ref_strides)
{
	// Grid-stride loop
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (long long dist_idx = index; dist_idx < dist_n; dist_idx += stride)
	{
		long i, j;
		i = calc_row_idx(dist_idx, ref_n);
		j = calc_col_idx(dist_idx, i, ref_n);
		if (j <= i)
		{
			continue;
		}
		
		// Set pointers to start of sketch i, j
		const uint64_t* ref_start = ref + i * ref_strides.sample_stride;
		const uint64_t* query_start = ref + j * ref_strides.sample_stride;

		float xsum = 0; float ysum = 0; float xysum = 0;
		float xsquaresum = 0; float ysquaresum = 0;
		for (int kmer_idx = 0; kmer_idx < kmer_n; ++kmer_idx)
		{
			// Get Jaccard distance and move pointers to next k-mer
			float jaccard_obs = jaccard_dist(ref_start, query_start, ref_strides, ref_strides); 
			ref_start += ref_strides.kmer_stride;
			query_start += ref_strides.kmer_stride;

			// Adjust for random matches
			float r1 = random_match[kmer_idx * ref_n + i];
			float r2 = random_match[kmer_idx * ref_n + j];
			float jaccard_expected = (r1 * r2) / (r1 + r2 - r1 * r2);
			float y = __logf(observed_excess(jaccard_obs, jaccard_expected, 1.0f));
			
			// Running totals for regression
			xsum += kmers[kmer_idx]; 
			ysum += y; 
			xysum += kmers[kmer_idx] * y;
			xsquaresum += kmers[kmer_idx] * kmers[kmer_idx];
			ysquaresum += y * y;
		}
		
		// Run the regression, and store results in dists
		simple_linear_regression(dists + dist_idx,
								 dists + dist_n + dist_idx,
								 xsum,
								 ysum,
								 xysum,
								 xsquaresum,
								 ysquaresum,
								 kmer_n);

		// Progress indicator
		// The >> 10 is a divide by 1024 - update roughly every 0.1%
		if (dist_idx % (dist_n >> 10) == 0) 
		{
			printf("%cProgress (GPU): %.1lf%%", 13, (float)dist_idx/dist_n * 100);
		}
	}
}

/***************
*			   *
*	Host code  *
*			   *	
***************/

// Turn a vector of references into a flattened vector of
// uint64 with strides bins * kmers * samples
thrust::host_vector<uint64_t> flatten_by_bins(
	const std::vector<Reference>& sketches,
	const std::vector<size_t>& kmer_lengths,
	SketchStrides& strides)
{
	// Set strides structure
	const size_t num_bins = strides.sketchsize64 * strides.bbits;
	assert(num_bins == sketches[0].get_sketch[kmer_lengths[0]].size());
	strides.bin_stride = 1;
	strides.kmer_stride = strides.bin_stride * num_bins;
	strides.sample_stride = strides.kmer_stride * kmer_lengths.size();
	
	// Iterate over each dimension to flatten
	thrust::host_vector<uint64_t> flat_ref(strides.sample_stride * sketches.size());
	auto flat_ref_it = flat_ref.begin();
	for (auto sample_it = sketches.cbegin(); sample_it != sketches.cend(); sample_it++)
	{
		for (auto kmer_it = kmer_lengths.cbegin(); kmer_it != kmer_lengths.cend(); kmer_it++)
		{
			thrust::copy(sample_it->get_sketch(*kmer_it).cbegin(),
						 sample_it->get_sketch(*kmer_it).cend(),
						 flat_ref_it);
            flat_ref_it += sample_it->get_sketch(*kmer_it).size();
		}
	}
	return flat_ref;
}

// Turn a vector of queries into a flattened vector of
// uint64 with strides samples * bins * kmers
thrust::host_vector<uint64_t> flatten_by_samples(
	const std::vector<Reference>& sketches,
	const std::vector<size_t>& kmer_lengths,
	SketchStrides& strides)
{
	// Set strides
	const size_t num_bins = strides.sketchsize64 * strides.bbits;
	assert(num_bins == sketches[0].get_sketch[kmer_lengths[0]].size());
	strides.sample_stride = 1;
	strides.bin_stride = sketches.size();
	strides.kmer_stride = strides.bin_stride * num_bins;

	// Stride by bins then restride by samples
	// This is 4x faster than striding by samples by looping over References vector, 
	// presumably because many fewer dereferences are being used
	SketchStrides old_strides = strides;
	thrust::host_vector<uint64_t> flat_bins = flatten_by_bins(sketches, kmer_lengths, old_strides);
	thrust::host_vector<uint64_t> flat_ref(strides.kmer_stride * kmer_lengths.size());
	auto flat_ref_it = flat_ref.begin();
	for (size_t kmer_idx = 0; kmer_idx < kmer_lengths.size(); kmer_idx++)
	{
		for (size_t bin_idx = 0; bin_idx < num_bins; bin_idx++)
		{
			for (size_t sample_idx = 0; sample_idx < sketches.size(); sample_idx++)
			{
				*flat_ref_it = flat_bins[sample_idx * old_strides.sample_stride + \
										 bin_idx * old_strides.bin_stride + \
										 kmer_idx * old_strides.kmer_stride];
				flat_ref_it++; 
			}
		}
	}

	return flat_ref;
}

// Calculates the random match probability for all sketches at all k-mer lengths
thrust::host_vector<float> preloadRandom(std::vector<Reference>& sketches, 
								 		 const std::vector<size_t>& kmer_lengths) {
	thrust::host_vector<float> random_sample_strided(sketches.size() * kmer_lengths.size());
	for (unsigned int sketch_idx = 0; sketch_idx < sketches.size(); sketch_idx++) {
		for (unsigned int kmer_idx = 0; kmer_idx < kmer_lengths.size(); kmer_idx++) {
			random_sample_strided[kmer_idx * sketches.size() + sketch_idx] = 
				(float)sketches[sketch_idx].random_match(kmer_lengths[kmer_idx]);
		}
	}
	return random_sample_strided;
}

DeviceMemory loadDeviceMemory(SketchStrides& ref_strides,
					  SketchStrides& query_strides,
					  std::vector<Reference>& ref_sketches,
					  std::vector<Reference>& query_sketches,
					  const SketchSlice& sample_slice,
					  const std::vector<size_t>& kmer_lengths,
					  long long dist_rows,
					  const bool self) {
	DeviceMemory loaded;

	// Need to (or easiest to) make temporary copies until we get
	// std::span in C++20

	// I think this use of pointers is not leaking memory - but 
	// should check whether new and unique_ptr is better
	std::unique_ptr<std::vector<Reference>> ref_subsample;
	if (sample_slice.ref_size < ref_sketches.size()) {
		ref_subsample.reset(new \
			std::vector<Reference>(ref_sketches.begin() + sample_slice.ref_offset,
								   ref_sketches.begin() + sample_slice.ref_offset + sample_slice.ref_size));
	} else {
		ref_subsample.reset(&ref_sketches);
	}

	std::unique_ptr<std::vector<Reference>> query_subsample;
	if (!self && sample_slice.query_size < query_sketches.size()) {
			query_subsample.reset(new \
				std::vector<Reference>(query_sketches.begin() + sample_slice.query_offset,
									   query_sketches.begin() + sample_slice.query_offset + sample_slice.query_size));
	} else {
		query_subsample.reset(&query_sketches);
	}

	// Set up reference sketches, flatten and copy to device
	thrust::host_vector<uint64_t> flat_ref = flatten_by_samples(*ref_subsample, kmer_lengths, ref_strides);
	loaded.ref_sketches = flat_ref;

	// If ref v query mode, also flatten query vector and copy to device
	if (!self)
	{
		thrust::host_vector<uint64_t> flat_query = flatten_by_bins(*query_subsample, kmer_lengths, query_strides);
		loaded.query_sketches = flat_query;
	}

	// Preload random match chances
	loaded.ref_random = preloadRandom(*ref_subsample, kmer_lengths);
	if (!self) {
		thrust::host_vector<float> query_random = preloadRandom(*query_subsample, kmer_lengths);
		loaded.query_random = query_random;
	}

	// Copy other arrays needed on device (kmers and distance output)
	loaded.kmers = kmer_lengths;
	loaded.dist_mat.resize(dist_rows*2, 0);

	return(loaded);
}

// Checks bbits, sketchsize and k-mer lengths are identical in
// all sketches
// throws runtime_error if mismatches (should be ensured in passing
// code)
void checkSketchParamsMatch(const std::vector<Reference>& sketches, 
	const std::vector<size_t>& kmer_lengths, 
	const size_t bbits, 
	const size_t sketchsize64)
{
	for (auto sketch_it = sketches.cbegin(); sketch_it != sketches.cend(); sketch_it++)
	{
		if (sketch_it->bbits() != bbits)
		{
			throw std::runtime_error("Mismatching bbits in sketches");
		}
		if (sketch_it->sketchsize64() != sketchsize64)
		{
			throw std::runtime_error("Mismatching sketchsize64 in sketches");
		}
		if (sketch_it->kmer_lengths() != kmer_lengths)
		{
			throw std::runtime_error("Mismatching k-mer lengths in sketches");
		}
	}
}

// Get the blockSize and blockCount for CUDA call
std::tuple<size_t, size_t> getBlockSize(const size_t ref_samples,
										const size_t query_samples,
									    const size_t dist_rows) {
	size_t blockSize, blockCount;
	if (query_samples > 0) {
		// Each block processes a single query. As max size is 512 threads
		// per block, may need multiple blocks (non-exact multiples lead
		// to some wasted computation in threads)
		// We take the next multiple of 32 that is larger than the number of
		// reference sketches, up to a maximum of 512
		blockSize = std::min(512, (int)(32 * (ref_samples + 32 - 1) / 32));
		size_t blocksPerQuery = (ref_samples + blockSize - 1) / blockSize;
		blockCount = blocksPerQuery * query_samples;
	} else {
		// Empirically a blockSize (selfBlockSize global const) of 32 or 256 seemed best
		blockSize = selfBlockSize;
		blockCount = (dist_rows + blockSize - 1) / blockSize;
	}
	return(std::make_tuple(blockSize, blockCount));
} 

// Run the distance calculations, reading/writing into device_arrays
// Cache preferences:
// Upper dist memory access is hard to predict, so try and cache as much
// as possible
// Query uses on-chip cache (__shared__) to store query sketch
// std::chrono::steady_clock::time_point b;
void dispatchDists(DeviceMemory& device_arrays,
				   std::vector<Reference>& ref_sketches,
				   std::vector<Reference>& query_sketches,
				   SketchStrides& ref_strides,
				   SketchStrides& query_strides,
				   const SketchSlice& sketch_subsample,
				   const std::vector<size_t>& kmer_lengths,
				   const bool self) {
	if (self) {
		// square 'self' block
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		
		long long chunk_dist_rows = static_cast<long long>(
										0.5*(sketch_subsample.ref_size)*(sketch_subsample.ref_size - 1));
		device_arrays = loadDeviceMemory(
			ref_strides,
			query_strides,
			ref_sketches,
			query_sketches,
			sketch_subsample,
			kmer_lengths,
			chunk_dist_rows,
			true);

		// hipDeviceSynchronize();	
		// b = std::chrono::steady_clock::now()

		size_t blockSize, blockCount;
		std::tie(blockSize, blockCount) = getBlockSize(sketch_subsample.ref_size, 
													   sketch_subsample.ref_size,
													   chunk_dist_rows);
		calculate_self_dists<<<blockCount, selfBlockSize>>>
			(
				thrust::raw_pointer_cast(&device_arrays.ref_sketches[0]),
				sketch_subsample.ref_size,
				thrust::raw_pointer_cast(&device_arrays.kmers[0]),
				kmer_lengths.size(),
				thrust::raw_pointer_cast(&device_arrays.dist_mat[0]),
				chunk_dist_rows,
				thrust::raw_pointer_cast(&device_arrays.ref_random[0]),
				ref_strides
			);
	} else {
		// 'query' block
		hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte); 
		hipDeviceSetCacheConfig(hipFuncCachePreferEqual);

		long long chunk_dist_rows = sketch_subsample.ref_size * sketch_subsample.query_size;
		device_arrays = loadDeviceMemory(
			ref_strides,
			query_strides,
			ref_sketches,
			query_sketches,
			sketch_subsample,
			kmer_lengths,
			chunk_dist_rows,
			false);
			
		size_t blockSize, blockCount;
		std::tie(blockSize, blockCount) = getBlockSize(sketch_subsample.ref_size, 
													   sketch_subsample.query_size,
													   chunk_dist_rows);
		
		// hipDeviceSynchronize();	
		// b = std::chrono::steady_clock::now()

		// Third argument is the size of __shared__ memory needed by a thread block
		// This is equal to the query sketch size in bytes (at a single k-mer length)
		calculate_query_dists<<<blockCount, blockSize, 
								query_strides.sketchsize64*query_strides.bbits*sizeof(uint64_t)>>>
		(
			thrust::raw_pointer_cast(&device_arrays.ref_sketches[0]),
			ref_sketches.size(),
			thrust::raw_pointer_cast(&device_arrays.query_sketches[0]),
			query_sketches.size(),
			thrust::raw_pointer_cast(&device_arrays.kmers[0]),
			kmer_lengths.size(),
			thrust::raw_pointer_cast(&device_arrays.dist_mat[0]),
			chunk_dist_rows,
			thrust::raw_pointer_cast(&device_arrays.ref_random[0]),
			thrust::raw_pointer_cast(&device_arrays.query_random[0]),
			ref_strides,
			query_strides
		);
	}
	printf("%cProgress (GPU): 100.0%%", 13);
	std::cout << std::endl << "" << std::endl;
}

// Main function callable via API
// Checks inputs
// Flattens sketches
// Copies flattened sketches to device
// Runs kernel function across distance elements
// Copies and returns results
NumpyMatrix query_db_cuda(std::vector<Reference>& ref_sketches,
	std::vector<Reference>& query_sketches,
	const std::vector<size_t>& kmer_lengths,
	const int device_id,
	const unsigned int num_cpu_threads)
{
	std::cerr << "Calculating distances on GPU device " << device_id << std::endl;
	
	// Initialise device
	hipSetDevice(device_id);
	hipDeviceReset();

	// Check sketches are compatible
	bool self = false;
	size_t bbits = ref_sketches[0].bbits();
	size_t sketchsize64 = ref_sketches[0].sketchsize64();
	checkSketchParamsMatch(ref_sketches, kmer_lengths, bbits, sketchsize64);
	
	// Set up sketch information and sizes
	SketchStrides ref_strides;
	ref_strides.bbits = bbits;
	ref_strides.sketchsize64 = sketchsize64;
	SketchStrides query_strides = ref_strides;

	long long dist_rows; long n_samples = 0;
	if (ref_sketches == query_sketches)
    {
		self = true;
		dist_rows = static_cast<long long>(0.5*(ref_sketches.size())*(ref_sketches.size() - 1));
		n_samples = ref_sketches.size(); 
	}
	else
	{
		// Also check query sketches are compatible
		checkSketchParamsMatch(query_sketches, kmer_lengths, bbits, sketchsize64);
		dist_rows = ref_sketches.size() * query_sketches.size();
		n_samples = ref_sketches.size() + query_sketches.size(); 
	}
	double est_size  = (bbits * sketchsize64 * kmer_lengths.size() * n_samples * sizeof(uint64_t) + \ // Size of sketches
						kmer_lengths.size() * n_samples * sizeof(float) + \                           // Size of random matches
						dist_rows * 2 * sizeof(float));												  // Size of distance matrix
	std::cerr << "Estimated device memory required: " << std::fixed << std::setprecision(0) << est_size/(1048576) << "Mb" << std::endl;

	size_t mem_free = 0; size_t mem_total = 0;
	hipMemGetInfo(&mem_free, &mem_total);
	std::cerr << "Total device memory: " << std::fixed << std::setprecision(0) << mem_total/(1048576) << "Mb" << std::endl;
	std::cerr << "Free device memory: " << std::fixed << std::setprecision(0) << mem_free/(1048576) << "Mb" << std::endl;

	if (est_size > mem_free * (1 - mem_epsilon) && !self) {
		throw std::runtime_error("Using greater than device memory is unsupported for query mode. "
							     "Split your input into smaller chunks");	
	}

	// Ready to run dists on device
	DeviceMemory device_arrays;
	SketchSlice sketch_subsample;
	unsigned int chunks = 1;
	std::vector<float> dist_results(dist_rows * 2);
	NumpyMatrix coreSquare, accessorySquare; 
	if (self)
	{
		// To prevent memory being exceeded, total distance matrix is split up into
		// chunks which do fit in memory. These are iterated over in the same order
		// as a square distance matrix. The i = j chunks are 'self', i < j can be skipped
		// as they contain only lower triangle values, i > j work as query vs ref
		chunks = floor(est_size / (mem_free * (1 - mem_epsilon))) + 1;
		size_t calc_per_chunk = n_samples / chunks;
		unsigned int num_big_chunks = n_samples % chunks;

		// Only allocate these square matrices if they are needed
		if (chunks > 1) {
			coreSquare.resize(n_samples, n_samples);
			accessorySquare.resize(n_samples, n_samples);
		}
		unsigned int total_chunks = (chunks * (chunks + 1)) >> 1;
		unsigned int chunk_count = 0;

		sketch_subsample.ref_offset = 0; 
		for (unsigned int chunk_i = 0; chunk_i < chunks; chunk_i++) {
			sketch_subsample.ref_size = calc_per_chunk;
			if (chunk_i < num_big_chunks) {
				sketch_subsample.ref_size++;
			}
			
			sketch_subsample.query_offset = sketch_subsample.ref_size; 
			for (unsigned int chunk_j = chunk_i; chunk_j < chunks; chunk_j++) {
				printf("Running chunk %ud of %ud\n", ++chunk_count, total_chunks);
				sketch_subsample.query_size = calc_per_chunk;
				if (chunk_j < num_big_chunks) {
					sketch_subsample.query_size++;
				}
				
				if (chunk_i == chunk_j) {
					// 'self' blocks
					dispatchDists(device_arrays,
						ref_sketches,
						ref_sketches,
						ref_strides,
						query_strides,
						sketch_subsample,
						kmer_lengths,
						true);
				} else {
					// 'query' block
					dispatchDists(device_arrays,
						ref_sketches,
						query_sketches,
						ref_strides,
						query_strides,
						sketch_subsample,
						kmer_lengths,
						false);
				}
				sketch_subsample.query_offset += sketch_subsample.query_size; 

				// Read intermediate dists out
				if (chunks > 1) {
					try {
						// Copy results from device into Nx2 matrix
						std::vector<float> block_results;
						thrust::copy(device_arrays.dist_mat.begin(), device_arrays.dist_mat.end(), block_results.begin());
						NumpyMatrix blockMat = \
							Eigen::Map<Eigen::Matrix<float,Eigen::Dynamic,2,Eigen::RowMajor> >(block_results.data(),block_results.size()/2,2);
						
						// Convert each long form column of Nx2 matrix into square distance matrix
						// Add this square matrix into the correct submatrix (block) of the final square matrix
						longToSquareBlock(coreSquare,
										  accessorySquare,
										  sketch_subsample,
										  block_results,
										  num_cpu_threads);

					} catch (thrust::system_error &e) {
						std::cerr << "Error getting result: " << std::endl;
						std::cerr << e.what() << std::endl;
						exit(1);
					}
					
				}

			}
			sketch_subsample.ref_offset += sketch_subsample.ref_size; 
		}

	}
	else
	{
		sketch_subsample.ref_size = ref_sketches.size();
		sketch_subsample.query_size = query_sketches.size();
		dispatchDists(device_arrays,
			ref_sketches,
			query_sketches,
			ref_strides,
			query_strides,
			sketch_subsample,
			kmer_lengths,
			false);	
	}
	// hipDeviceSynchronize();
	// std::chrono::steady_clock::time_point c = std::chrono::steady_clock::now();
	
	// copy results from device back to host
	// try and keep Eigen code in .cpp files (http://eigen.tuxfamily.org/dox-devel///TopicCUDA.html)
	NumpyMatrix dists_ret_matrix;
	if (self && chunks > 1) {
		// Chunked computation yields square matrix, which needs to be converted back to long
		// form
		dists_ret_matrix = twoColumnSquareToLong(coreSquare,
												 accessorySquare,
												 num_cpu_threads);
	} else {
		try {
			// Single chunks just need to be moved from the device into the return vector
			// CUDA code now returns column major data (i.e. all core dists, then all accessory dists)
			// to try and coalesce writes.
			// NB: almost all other code is row major (i.e. sample core then accessory, then next sample)
			thrust::copy(device_arrays.dist_mat.begin(), device_arrays.dist_mat.end(), dist_results.begin());
			dists_ret_matrix = \
				Eigen::Map<Eigen::Matrix<float,Eigen::Dynamic,2,Eigen::RowMajor> >(dist_results.data(),dist_results.size()/2,2);
		} catch (thrust::system_error &e) {
			// output a non-threatening but likely inaccurate error message and exit
			// e.g. 'trivial_device_copy D->H failed: unspecified launch failure'
			// error will have occurred elsewhere as launch is async, but better to catch 
			// and deal with it here
			std::cerr << "Error getting result: " << std::endl;
			std::cerr << e.what() << std::endl;
			exit(1);
		}
	}

	/* Code used to time in development:
	// Report timings of each step
	std::chrono::steady_clock::time_point d = std::chrono::steady_clock::now();
	std::chrono::duration<double> load_time = std::chrono::duration_cast<std::chrono::duration<double> >(b-a);
	std::chrono::duration<double> calc_time = std::chrono::duration_cast<std::chrono::duration<double> >(c-b);
	std::chrono::duration<double> save_time = std::chrono::duration_cast<std::chrono::duration<double> >(d-c);

	std::cout << "Loading: " << load_time.count()<< "s" << std::endl;
	std::cout << "Distances: " << calc_time.count()<< "s" << std::endl;
	std::cout << "Saving: " << save_time.count()<< "s" << std::endl;
	*/

	return dists_ret_matrix;
}