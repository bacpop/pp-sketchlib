#include "hip/hip_runtime.h"

/*
 *
 * sketch.cu
 * CUDA version of bindash sketch method
 *
 */

#include <stdint.h>

#include <pybind11/pybind11.h>
namespace py = pybind11;

// memcpy_async
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#pragma diag_suppress static_var_with_dynamic_init

#include "cuda.cuh"
#include "gpu.hpp"

// nthash
#include "sketch/nthash_tables.hpp"

// Tables on device
__constant__ uint64_t d_seedTab[256];
__constant__ uint64_t d_A33r[33];
__constant__ uint64_t d_A31l[31];
__constant__ uint64_t d_C33r[33];
__constant__ uint64_t d_C31l[31];
__constant__ uint64_t d_G33r[33];
__constant__ uint64_t d_G31l[31];
__constant__ uint64_t d_T33r[33];
__constant__ uint64_t d_T31l[31];
__constant__ uint64_t d_N33r[33];
__constant__ uint64_t d_N31l[31];
__constant__ uint64_t *d_msTab33r[256];
__constant__ uint64_t *d_msTab31l[256];

// main nthash functions - see nthash.hpp
// All others are built from calling these

__device__ inline uint64_t rol1(const uint64_t v) {
  return (v << 1) | (v >> 63);
}

__device__ inline uint64_t ror1(const uint64_t v) {
  return (v >> 1) | (v << 63);
}

__device__ inline uint64_t rol31(const uint64_t v, unsigned s) {
  s %= 31;
  return ((v << s) | (v >> (31 - s))) & 0x7FFFFFFF;
}

__device__ inline uint64_t rol33(const uint64_t v, unsigned s) {
  s %= 33;
  return ((v << s) | (v >> (33 - s))) & 0x1FFFFFFFF;
}

__device__ inline uint64_t swapbits033(const uint64_t v) {
  uint64_t x = (v ^ (v >> 33)) & 1;
  return v ^ (x | (x << 33));
}

__device__ inline uint64_t swapbits3263(const uint64_t v) {
  uint64_t x = ((v >> 32) ^ (v >> 63)) & 1;
  return v ^ ((x << 32) | (x << 63));
}

// Forward strand hash for first k-mer
__device__ inline void NT64(const char *kmerSeq, const unsigned k,
                            uint64_t &fhVal) {
  fhVal = 0;
  for (int i = k - 1; i >= 0; i--) {
    // Ns are removed, but this is how to check for them
    /*
    if(seedTab[(unsigned char)kmerSeq[i * baseStride]]==seedN) {
        locN=i;
        return false;
    }
    */
    fhVal = rol1(fhVal);
    fhVal = swapbits033(fhVal);
    fhVal ^= d_seedTab[(unsigned char)kmerSeq[k - 1 - i]];
  }
  // return true;
}

// Both strand hashes for first k-mer
__device__ inline void NTC64(const char *kmerSeq, const unsigned k,
                             uint64_t &fhVal, uint64_t &rhVal, uint64_t &hVal) {
  hVal = fhVal = rhVal = 0;
  for (int i = (k - 1); i >= 0; i--) {
    // Ns are removed, but this is how to check for them
    /*
    if(seedTab[(unsigned char)kmerSeq[i * baseStride]]==seedN) {
        locN = i;
        return false;
    }
    */
    fhVal = rol1(fhVal);
    fhVal = swapbits033(fhVal);
    fhVal ^= d_seedTab[(unsigned char)kmerSeq[k - 1 - i]];

    rhVal = rol1(rhVal);
    rhVal = swapbits033(rhVal);
    rhVal ^= d_seedTab[(unsigned char)kmerSeq[i] & cpOff];
  }
  hVal = (rhVal < fhVal) ? rhVal : fhVal;
  // return true;
}

// forward-strand ntHash for subsequent sliding k-mers
__device__ inline uint64_t NTF64(const uint64_t fhVal, const unsigned k,
                                 const unsigned char charOut,
                                 const unsigned char charIn) {
  uint64_t hVal = rol1(fhVal);
  hVal = swapbits033(hVal);
  hVal ^= d_seedTab[charIn];
  hVal ^= (d_msTab31l[charOut][k % 31] | d_msTab33r[charOut][k % 33]);
  return hVal;
}

// reverse-complement ntHash for subsequent sliding k-mers
__device__ inline uint64_t NTR64(const uint64_t rhVal, const unsigned k,
                                 const unsigned char charOut,
                                 const unsigned char charIn) {
  uint64_t hVal = rhVal ^ (d_msTab31l[charIn & cpOff][k % 31] |
                           d_msTab33r[charIn & cpOff][k % 33]);
  hVal ^= d_seedTab[charOut & cpOff];
  hVal = ror1(hVal);
  hVal = swapbits3263(hVal);
  return hVal;
}

// Create a new hash from an nthash
__device__ inline uint64_t shifthash(const uint64_t hVal, const unsigned k,
                                     const unsigned i) {
  uint64_t tVal = hVal * (i ^ k * multiSeed);
  tVal ^= tVal >> multiShift;
  return (tVal);
}

// parameters - these are currently hard coded based on a 3090 (24Gb RAM)
const unsigned int table_width_bits = 30; // 2^30 + 1 = 1073741825 =~ 1 billion k-mers
constexpr uint64_t table_width{0x3FFFFFFF};       // 30 lowest bits ON
const int hash_per_hash =
    2; // This should be 2, or the table is likely too narrow
const int table_rows =
    4; // Number of hashes, should be a multiple of hash_per_hash
constexpr uint64_t table_cells = table_rows * table_width;

// Countmin
// See countmin.cpp
GPUCountMin::GPUCountMin()
    : _table_width_bits(table_width_bits),
      _table_width(table_width), _hash_per_hash(hash_per_hash),
      _table_rows(table_rows), _table_cells(table_cells) {
  CUDA_CALL(hipMalloc((void **)&_d_countmin_table,
                       table_cells * sizeof(unsigned int)));
  reset();
}

GPUCountMin::~GPUCountMin() { CUDA_CALL(hipFree(_d_countmin_table)); }

// Loop variables are global constants defined in gpu.hpp
__device__ unsigned int add_count_min(unsigned int *d_countmin_table,
                                      uint64_t hash_val, const int k) {
  unsigned int min_count = UINT32_MAX;
  for (int hash_nr = 0; hash_nr < table_rows; hash_nr += hash_per_hash) {
    uint64_t current_hash = hash_val;
    for (uint i = 0; i < hash_per_hash; i++) {
      uint32_t hash_val_masked = current_hash & table_width;
      unsigned int cell_count =
          atomicInc(d_countmin_table + (hash_nr + i) * table_width +
                        hash_val_masked,
                    UINT32_MAX) +
          1;
      if (cell_count < min_count) {
        min_count = cell_count;
      }
      current_hash = current_hash >> table_width_bits;
    }
    hash_val = shifthash(hash_val, k, hash_nr / 2);
  }
  return (min_count);
}

void GPUCountMin::reset() {
  CUDA_CALL(
      hipMemset(_d_countmin_table, 0, table_cells * sizeof(unsigned int)));
}

// bindash functions
const uint64_t SIGN_MOD = (1ULL << 61ULL) - 1ULL;

// countmin and binsign
__device__ void binhash(uint64_t *signs, unsigned int *countmin_table,
                        const uint64_t hash, const uint64_t binsize,
                        const int k, const uint16_t min_count) {
  uint64_t sign = hash % SIGN_MOD;
  uint64_t binidx = sign / binsize;
  // printf("binidx:%llu sign:%llu\n", binidx, sign);

  // Only consider if the bin is yet to be filled, or is min in bin
  if (signs[binidx] == UINT64_MAX || sign < signs[binidx]) {
    if (add_count_min(countmin_table, hash, k) >= min_count) {
      signs[binidx] = sign;
    }
  }
  __syncwarp();
}

// hash iterator object
__global__ void process_reads(char *read_seq,
                              const size_t n_reads,
                              const size_t read_length,
                              const int k,
                              uint64_t *signs,
                              const uint64_t binsize,
                              unsigned int *countmin_table,
                              const bool use_rc,
                              const uint16_t min_count) {
  // Load reads in block into shared memory
  extern __shared__ char read_shared[];
  auto block = cooperative_groups::this_thread_block();
  cooperative_groups::memcpy_async(
    block,
    &read_shared,
    read_seq + read_length * (blockIdx.x * blockDim.x),
    sizeof(char) * read_length * blockDim.x);
  cooperative_groups::wait(block);

  int read_index = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t fhVal, rhVal, hVal;
  if (read_index < n_reads) {
    // Get first valid k-mer
    if (use_rc) {
      NTC64(read_shared + threadIdx.x * read_length, k, fhVal, rhVal, hVal);
      binhash(signs, countmin_table, hVal, binsize, k, min_count);
    } else {
      NT64(read_shared + threadIdx.x * read_length, k, fhVal);
      binhash(signs, countmin_table, hVal, binsize, k, min_count);
    }

    // Roll through remaining k-mers in the read
    for (int pos = 0; pos < read_length - k; pos++) {
      fhVal = NTF64(fhVal, k, read_shared[threadIdx.x * read_length + pos],
                    read_shared[threadIdx.x * read_length + pos + k]);
      if (use_rc) {
        rhVal = NTR64(rhVal, k, read_shared[threadIdx.x * read_length + pos],
                      read_shared[threadIdx.x * read_length + pos + k]);
        hVal = (rhVal < fhVal) ? rhVal : fhVal;
        binhash(signs, countmin_table, hVal, binsize, k, min_count);
      } else {
        binhash(signs, countmin_table, fhVal, binsize, k, min_count);
      }
    }
  }
  __syncwarp();
}

DeviceReads::DeviceReads(const SeqBuf &seq_in, const size_t n_threads)
    : seq(make_unique<SeqBuf>(seq_in)),
      n_reads(seq_in.n_full_seqs()), read_length(seq_in.max_length()),
      current_block(0), buffer_filled(0) {

  // Set up buffer to load in reads (on host)
  size_t mem_free = 0;
  size_t mem_total = 0;
  CUDA_CALL(hipMemGetInfo(&mem_free, &mem_total));
  buffer_size = (mem_free * 0.9) / (read_length * sizeof(char));
  buffer_blocks = std::floor(n_reads / (static_cast<double>(buffer_size) + 1)) + 1;
  if (buffer_size > n_reads) {
    buffer_size = n_reads;
    buffer_blocks = 1;
  }
  host_buffer.resize(buffer_size * read_length);
  CUDA_CALL(hipHostRegister(
              host_buffer.data(),
              host_buffer.size() * sizeof(char),
              hipHostRegisterDefault));

  // Buffer to store reads (on device)
  CUDA_CALL(hipMalloc((void **)&d_reads,
                        buffer_size * read_length * sizeof(char)));

  CUDA_CALL(hipStreamCreate(&memory_stream));
}

DeviceReads::~DeviceReads() {
  CUDA_CALL(hipHostUnregister(host_buffer.data()));
  CUDA_CALL(hipFree(d_reads));
  CUDA_CALL(hipStreamDestroy(memory_stream));
}

bool DeviceReads::next_buffer() {
  bool success;
  if (current_block < buffer_blocks) {
    size_t start = current_block * buffer_size;
    size_t end = (current_block + 1) * buffer_size;
    if (end > seq->n_full_seqs()) {
      end = seq->n_full_seqs();
    }
    buffer_filled = end - start;

    seq->load_seqs(host_buffer, start, end);
    CUDA_CALL(hipMemcpyAsync(d_reads,
                              host_buffer.data(),
                              buffer_filled * read_length * sizeof(char),
                              hipMemcpyDefault,
                              memory_stream));

    current_block++;
    success = true;
  } else {
    buffer_filled = 0;
    success = false;
  }
  return success;
}

void copyNtHashTablesToDevice() {
  CUDA_CALL(
      hipMemcpyToSymbolAsync(HIP_SYMBOL(d_seedTab), seedTab, 256 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_A33r), A33r, 33 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_A31l), A31l, 31 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_C33r), C33r, 33 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_C31l), C31l, 31 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_G33r), G33r, 33 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_G31l), G31l, 31 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_T33r), T33r, 33 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_T31l), T31l, 31 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_N33r), N33r, 33 * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_N31l), N31l, 31 * sizeof(uint64_t)));
  CUDA_CALL(hipDeviceSynchronize());

  uint64_t *A33r_ptr, *A31l_ptr, *C33r_ptr, *C31l_ptr, *G33r_ptr, *G31l_ptr,
      *T33r_ptr, *T31l_ptr, *N33r_ptr, *N31l_ptr;
  CUDA_CALL(hipGetSymbolAddress((void **)&A33r_ptr, d_A33r));
  CUDA_CALL(hipGetSymbolAddress((void **)&A31l_ptr, d_A31l));
  CUDA_CALL(hipGetSymbolAddress((void **)&C33r_ptr, d_C33r));
  CUDA_CALL(hipGetSymbolAddress((void **)&C31l_ptr, d_C31l));
  CUDA_CALL(hipGetSymbolAddress((void **)&G33r_ptr, d_G33r));
  CUDA_CALL(hipGetSymbolAddress((void **)&G31l_ptr, d_G31l));
  CUDA_CALL(hipGetSymbolAddress((void **)&T33r_ptr, d_T33r));
  CUDA_CALL(hipGetSymbolAddress((void **)&T31l_ptr, d_T31l));
  CUDA_CALL(hipGetSymbolAddress((void **)&N33r_ptr, d_N33r));
  CUDA_CALL(hipGetSymbolAddress((void **)&N31l_ptr, d_N31l));

  static const uint64_t *d_addr_msTab33r[256] = {
      N33r_ptr, T33r_ptr, N33r_ptr, G33r_ptr,
      A33r_ptr, A33r_ptr, N33r_ptr, C33r_ptr, // 0..7
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 8..15
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 16..23
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 24..31
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 32..39
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 40..47
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 48..55
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 56..63
      N33r_ptr, A33r_ptr, N33r_ptr, C33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, G33r_ptr, // 64..71
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 72..79
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      T33r_ptr, T33r_ptr, N33r_ptr, N33r_ptr, // 80..87
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 88..95
      N33r_ptr, A33r_ptr, N33r_ptr, C33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, G33r_ptr, // 96..103
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 104..111
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      T33r_ptr, T33r_ptr, N33r_ptr, N33r_ptr, // 112..119
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 120..127
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 128..135
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 136..143
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 144..151
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 152..159
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 160..167
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 168..175
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 176..183
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 184..191
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 192..199
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 200..207
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 208..215
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 216..223
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 224..231
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 232..239
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr, // 240..247
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr,
      N33r_ptr, N33r_ptr, N33r_ptr, N33r_ptr // 248..255
  };

  static const uint64_t *d_addr_msTab31l[256] = {
      N31l_ptr, T31l_ptr, N31l_ptr, G31l_ptr,
      A31l_ptr, A31l_ptr, N31l_ptr, C31l_ptr, // 0..7
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 8..15
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 16..23
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 24..31
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 32..39
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 40..47
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 48..55
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 56..63
      N31l_ptr, A31l_ptr, N31l_ptr, C31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, G31l_ptr, // 64..71
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 72..79
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      T31l_ptr, T31l_ptr, N31l_ptr, N31l_ptr, // 80..87
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 88..95
      N31l_ptr, A31l_ptr, N31l_ptr, C31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, G31l_ptr, // 96..103
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 104..111
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      T31l_ptr, T31l_ptr, N31l_ptr, N31l_ptr, // 112..119
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 120..127
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 128..135
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 136..143
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 144..151
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 152..159
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 160..167
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 168..175
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 176..183
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 184..191
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 192..199
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 200..207
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 208..215
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 216..223
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 224..231
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 232..239
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr, // 240..247
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr,
      N31l_ptr, N31l_ptr, N31l_ptr, N31l_ptr // 248..255
  };

  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_msTab31l), d_addr_msTab31l,
                                    256 * sizeof(uint64_t *)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_msTab33r), d_addr_msTab33r,
                                    256 * sizeof(uint64_t *)));
  CUDA_CALL(hipDeviceSynchronize());
}

// main function called here returns signs vector - rest can be done by
// sketch.cpp
std::vector<uint64_t>
get_signs(DeviceReads &reads,
          GPUCountMin &countmin, const int k, const bool use_rc,
          const uint16_t min_count, const uint64_t binsize,
          const uint64_t nbins) {
  // Set countmin to zero (already on device)
  countmin.reset();

  // Signs
  std::vector<uint64_t> signs(nbins, UINT64_MAX);
  uint64_t *d_signs;
  CUDA_CALL(hipMalloc((void **)&d_signs, nbins * sizeof(uint64_t)));
  CUDA_CALL(hipMemcpy(d_signs, signs.data(), nbins * sizeof(uint64_t),
                       hipMemcpyDefault));

  // Run process_read kernel, looping over reads loaded into buffer
  //      This runs nthash on read sequence at all k-mer lengths
  //      Check vs signs and countmin on whether to add each
  const size_t blockSize = 64;
  while (reads.next_buffer()) {
    size_t blockCount = (reads.buffer_count() + blockSize - 1) / blockSize;
    process_reads<<<blockCount,
                  blockSize,
                  reads.length() * blockSize * sizeof(char),
                  reads.stream()>>>(
      reads.read_ptr(),
      reads.buffer_count(),
      reads.length(),
      k,
      d_signs,
      binsize,
      countmin.get_table(),
      use_rc,
      min_count
    );
    CUDA_CALL(hipGetLastError());

    // Check for interrupt
    if (PyErr_CheckSignals() != 0) {
      throw py::error_already_set();
    }
  }

  // Copy signs back from device
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(signs.data(), d_signs, nbins * sizeof(uint64_t),
                       hipMemcpyDefault));
  CUDA_CALL(hipFree(d_signs));

  fprintf(stderr, "%ck = %d  ", 13, k);

  return (signs);
}
