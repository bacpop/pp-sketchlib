#include "hip/hip_runtime.h"

#include "cuda.cuh"
#include "device_reads.cuh"

DeviceReads::DeviceReads(const std::shared_ptr<SeqBuf> &seq_ptr,
                         const size_t n_threads)
    : seq(seq_ptr), n_reads(seq->n_full_seqs()), read_length(seq->max_length()),
      current_block(0), buffer_filled(0), loaded_first(false) {
  // Set up buffer to load in reads (on host)
  size_t mem_free = 0;
  size_t mem_total = 0;
  CUDA_CALL(hipMemGetInfo(&mem_free, &mem_total));
  buffer_size = (mem_free * 0.9) / (read_length * sizeof(char));
  buffer_blocks =
      std::floor(n_reads / (static_cast<double>(buffer_size) + 1)) + 1;
  if (buffer_size > n_reads) {
    buffer_size = n_reads;
    buffer_blocks = 1;
  }
  host_buffer.resize(buffer_size * read_length);
  CUDA_CALL_NOTHROW(hipHostRegister(host_buffer.data(),
                                     host_buffer.size() * sizeof(char),
                                     hipHostRegisterDefault));

  // Buffer to store reads (on device)
  CUDA_CALL(
      hipMalloc((void **)&d_reads, buffer_size * read_length * sizeof(char)));
}

DeviceReads::~DeviceReads() {
  CUDA_CALL_NOTHROW(hipHostUnregister(host_buffer.data()));
  CUDA_CALL_NOTHROW(hipFree(d_reads));
}

bool DeviceReads::next_buffer() {
  bool success;
  if (current_block < buffer_blocks) {
    if (buffer_blocks > 1 || !loaded_first) {
      size_t start = current_block * buffer_size;
      size_t end = (current_block + 1) * buffer_size;
      if (end > seq->n_full_seqs()) {
        end = seq->n_full_seqs();
      }
      buffer_filled = end - start;

      seq->load_seqs(host_buffer, start, end);
      CUDA_CALL(hipMemcpyAsync(d_reads, host_buffer.data(),
                                buffer_filled * read_length * sizeof(char),
                                hipMemcpyDefault));
      loaded_first = true;
    }
    current_block++;
    success = true;
  } else {
    success = false;
  }
  return success;
}
