#include "hip/hip_runtime.h"
/*
 *
 * dist.cu
 * PopPUNK dists using CUDA
 * nvcc compiled part (try to avoid eigen)
 *
 */

// std
#include <algorithm>
#include <assert.h>
#include <cfloat>
#include <cmath>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <stdexcept>
#include <stdlib.h>
#include <tuple>
#include <unistd.h>
#include <vector>

#include <pybind11/pybind11.h>
namespace py = pybind11;

// memcpy_async
#include <hipcub/hipcub.hpp>
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#pragma nv_diag_suppress static_var_with_dynamic_init

// internal headers
#include "cuda.cuh"
#include "containers.cuh"
#include "dist/matrix_idx.hpp"
#include "gpu.hpp"
#include "sketch/bitfuncs.hpp"

/******************
 *			          *
 *	Device code   *
 *			          *
 *******************/

// Ternary used in observed_excess
template <class T> __device__ T non_neg_minus(T a, T b) {
  return a > b ? (a - b) : 0;
}

// Calculates excess observations above a random value
template <class T> __device__ T observed_excess(T obs, T exp, T max) {
  T diff = non_neg_minus(obs, exp);
  return (diff * max / (max - exp));
}

// CUDA version of bindash dist function (see dist.cpp)
__device__ float jaccard_dist(const uint64_t *sketch1, const uint64_t *sketch2,
                              const size_t sketchsize64, const size_t bbits,
                              const size_t s1_stride, const size_t s2_stride) {
  size_t samebits = 0;
  for (int i = 0; i < sketchsize64; i++) {
    int bin_index = i * bbits;
    uint64_t bits = ~((uint64_t)0ULL);
    for (int j = 0; j < bbits; j++) {
      // Almost all kernel time is spent on this line
      // (bbits * sketchsize64 * N^2 * 2 8-byte memory loads)
      bits &=
          ~(sketch1[bin_index * s1_stride] ^ sketch2[bin_index * s2_stride]);
      bin_index++;
    }
    samebits += __popcll(bits); // CUDA 64-bit popcnt
  }

  const size_t maxnbits = sketchsize64 * NBITS(uint64_t);
  const size_t expected_samebits = (maxnbits >> bbits);
  size_t intersize = samebits;
  if (!expected_samebits) {
    size_t ret = observed_excess(samebits, expected_samebits, maxnbits);
  }

  size_t unionsize = NBITS(uint64_t) * sketchsize64;
  float jaccard = __fdiv_ru(intersize, unionsize);
  return (jaccard);
}

// Simple linear regression, exact solution
// Avoids use of dynamic memory allocation on device, or
// linear algebra libraries
__device__ void simple_linear_regression(float dists[],
                                         const float xsum, const float ysum,
                                         const float xysum,
                                         const float xsquaresum,
                                         const float ysquaresum, const int n,
                                         const int dist_col) {
  if (n < 2) {
    dists[0] = 0.0f;
    dists[1] = 0.0f;
  } else {
    // CUDA fast-math intrinsics on floats, which give comparable accuracy
    // Speed gain is fairly minimal, as most time spent on Jaccard distance
    // __fmul_ru(x, y) = x * y and rounds up.
    // __fpow(x, a) = x^a give 0 for x<0, so not using here (and it is slow)
    float xbar = xsum / n;
    float ybar = ysum / n;
    float x_diff = xsquaresum - __fmul_ru(xsum, xsum) / n;
    float y_diff = ysquaresum - __fmul_ru(ysum, ysum) / n;
    float xstddev = __fsqrt_ru((xsquaresum - __fmul_ru(xsum, xsum) / n) / n);
    float ystddev = __fsqrt_ru((ysquaresum - __fmul_ru(ysum, ysum) / n) / n);
    float r =
        __fdiv_ru(xysum - __fmul_ru(xsum, ysum) / n, __fsqrt_ru(x_diff * y_diff));
    float beta = __fmul_ru(r, __fdiv_ru(ystddev, xstddev));
    float alpha = __fmaf_ru(-beta, xbar, ybar); // maf: x * y + z

    // Store core/accessory in dists, truncating at zero
    // Memory should be initialised to zero so else block not strictly
    // necessary, but better safe than sorry!
    if (beta < 0.0f) {
      dists[0] = 1.0f - __expf(beta);
    } else {
      dists[0] = 0.0f;
    }

    if (alpha < 0.0f) {
      dists[1] = 1.0f - __expf(alpha);
    } else {
      dists[1] = 0.0f;
    }
  }
}

/******************
 *			      *
 *	Global code   *
 *			      *
 *******************/

// Main kernel functions run on the device,
// but callable from the host

__global__ void set_idx(long* idx, size_t row_samples, size_t col_samples, size_t col_offset) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < row_samples * col_samples;
    i += blockDim.x * gridDim.x) {
    idx[i] = col_offset + i % col_samples;
  }
}

__global__ void copy_top_k(float* sorted_dists, long* sorted_idx,
  float* all_sorted_dists, long* all_sorted_idx, int segment_size, int n_out,
    int kNN, int sketch_block_idx, int n_chunks, bool second_sort) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_out;
    i += blockDim.x * gridDim.x) {
    const int offset_in = (i / kNN) * segment_size + i % kNN;
    int offset_out = i;
    // If copying from the sorted kNN * n_chunk list into the final sparse matrix
    if (!second_sort) {
      // If copying from the sorted n_chunk dense list into the kNN * n_chunk list
      offset_out += ((i / kNN) * (kNN * (n_chunks - 1))) + (kNN * sketch_block_idx);
    }
    all_sorted_dists[offset_out] = sorted_dists[offset_in];
    all_sorted_idx[offset_out] = sorted_idx[offset_in];
  }
}

__global__ void calculate_dists(
    const bool self, const uint64_t *ref, const long ref_n,
    const uint64_t *query, const long query_n, const int *kmers,
    const int kmer_n, float *dists, const long long dist_n,
    const float *random_table, const uint16_t *ref_idx_lookup,
    const uint16_t *query_idx_lookup, const SketchStrides ref_strides,
    const SketchStrides query_strides, const RandomStrides random_strides,
    progress_ptrs progress, const bool use_shared,
    const int dist_col, const bool max_diagonal) {
  // Calculate indices for query, ref and results
  int ref_idx, query_idx, dist_idx;
  if (self) {
    // Blocks have the same i -- calculate blocks needed by each row up
    // to this point (blockIdx.x)
    int blocksDone = 0;
    for (query_idx = 0; query_idx < ref_n; query_idx++) {
      blocksDone += (ref_n + blockDim.x - 2 - query_idx) / blockDim.x;
      if (blocksDone > blockIdx.x) {
        break;
      }
    }
    // j (column) is given by multiplying the blocks needed for this i (row)
    // by the block size, plus offsets of i + 1 and the thread index
    int blocksPerQuery = (ref_n + blockDim.x - 2 - query_idx) / blockDim.x;
    ref_idx = query_idx + 1 + threadIdx.x +
              (blockIdx.x - (blocksDone - blocksPerQuery)) * blockDim.x;

    if (ref_idx < ref_n) {
      // Order of ref/query reversed here to give correct output order
      dist_idx = square_to_condensed(query_idx, ref_idx, ref_n);
    }
  } else {
    int blocksPerQuery = (ref_n + blockDim.x - 1) / blockDim.x;
    query_idx = blockIdx.x / blocksPerQuery;
    ref_idx = (blockIdx.x % blocksPerQuery) * blockDim.x + threadIdx.x;
    dist_idx = query_idx * ref_n + ref_idx;
  }
  __syncwarp();

  const uint64_t *ref_start = ref + ref_idx * ref_strides.sample_stride;
  const uint64_t *query_start = query + query_idx * query_strides.sample_stride;
  const float tolerance =
      __fdividef(5.0f, __int2float_rz(64 * ref_strides.sketchsize64));

  // Calculate Jaccard distances over k-mer lengths
  int kmer_used = 0;
  float xsum = 0.0f;
  float ysum = 0.0f;
  float xysum = 0.0f;
  float xsquaresum = 0.0f;
  float ysquaresum = 0.0f;
  bool stop = false;
  for (int kmer_idx = 0; kmer_idx < kmer_n; kmer_idx++) {
    // Copy query sketch into __shared__ mem
    // Uses all threads *in a single warp* to do the copy
    // NB there is no disadvantage vs using multiple warps, as they would have
    // to wait (see
    // https://stackoverflow.com/questions/15468059/copy-to-the-shared-memory-in-cuda)
    // NB for query these reads will be coalesced, but for ref they won't, as
    // can't coalesce both here (bin inner stride) and in jaccard (sample inner
    // stride)
    const uint64_t *query_ptr;
    extern __shared__ uint64_t query_shared[];
    int query_bin_strides;
    auto block = cooperative_groups::this_thread_block();
    __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
    if (block.thread_rank() == 0) {
      init(&barrier, block.size()); // Friend function initializes barrier
    }
    block.sync();
    if (use_shared) {
      size_t sketch_bins = query_strides.bbits * query_strides.sketchsize64;
      size_t sketch_stride = query_strides.bin_stride;
      if (threadIdx.x < warp_size) {
        for (int lidx = threadIdx.x; lidx < sketch_bins; lidx += warp_size) {
          cuda::memcpy_async(query_shared + lidx,
                             query_start + (lidx * sketch_stride),
                             sizeof(uint64_t),
                             barrier);
        }
      }
      query_ptr = query_shared;
      query_bin_strides = 1;
    } else {
      query_ptr = query_start;
      query_bin_strides = query_strides.bin_stride;
    }
    barrier.arrive_and_wait();

    // Some threads at the end of the last block will have nothing to do
    // Need to have conditional here to avoid block on __syncthreads() above
    if (ref_idx < ref_n) {
      // Calculate Jaccard distance at current k-mer length
      float jaccard_obs = jaccard_dist(
          ref_start, query_ptr, ref_strides.sketchsize64, ref_strides.bbits,
          ref_strides.bin_stride, query_bin_strides);

      // Adjust for random matches
      float jaccard_expected =
          random_table[kmer_idx * random_strides.kmer_stride +
                       ref_idx_lookup[ref_idx] *
                           random_strides.cluster_inner_stride +
                       query_idx_lookup[query_idx] *
                           random_strides.cluster_outer_stride];
      float jaccard = observed_excess(jaccard_obs, jaccard_expected, 1.0f);
      // Stop regression if distances =~ 0
      if (jaccard < tolerance) {
        // Would normally break here, but gives no advantage on a GPU as causes
        // warp to diverge
        // As the thread blocks are used to load the query in, adding a break
        // would actually cause a stall. So just stop adding
        stop = true;
      } else if (!stop) {
        float y = __logf(jaccard);
        // printf("i:%d j:%d k:%d r:%f jac:%f y:%f\n", ref_idx, query_idx,
        // kmer_idx, jaccard_expected, jaccard_obs, y);

        // Running totals for regression
        kmer_used++;
        int kmer = kmers[kmer_idx];
        xsum += kmer;
        ysum += y;
        xysum += kmer * y;
        xsquaresum += kmer * kmer;
        ysquaresum += y * y;
      }
    }

    // Move to next k-mer length
    ref_start += ref_strides.kmer_stride;
    query_start += query_strides.kmer_stride;
  }

  if (ref_idx < ref_n) {
    // Run the regression, and store results in dists
    float fitted_dists[2];
    // Set diagonal if you wish to ignore diagonals (set them to max)
    if (max_diagonal && ref_idx == query_idx) {
      fitted_dists[0] = INFINITY;
      fitted_dists[1] = INFINITY;
    } else {
      simple_linear_regression(fitted_dists, xsum, ysum, xysum, xsquaresum,
                               ysquaresum, kmer_used, dist_col);
    }
    if (dist_col < 0) {
      dists[dist_idx] = fitted_dists[0];
      dists[dist_idx + dist_n] = fitted_dists[1];
    } else {
      dists[dist_idx] = fitted_dists[dist_col];
    }

    update_progress(dist_idx, dist_n, progress);
  }
}

/***************
 *			       *
 *	Host code  *
 *			       *
 ***************/

// Get the blockSize and blockCount for CUDA call
std::tuple<size_t, size_t> getBlockSize(const size_t ref_samples,
                                        const size_t query_samples,
                                        const size_t dist_rows,
                                        const bool self) {
  // Each block processes a single query. As max size is 512 threads
  // per block, may need multiple blocks (non-exact multiples lead
  // to some wasted computation in threads)
  // We take the next multiple of 32 that is larger than the number of
  // reference sketches, up to a maximum of 512
  size_t blockSize =
      std::min(256, 32 * static_cast<int>((ref_samples + 32 - 1) / 32));
  size_t blockCount = 0;
  if (self) {
    for (int i = 0; i < ref_samples; i++) {
      blockCount += (ref_samples + blockSize - 2 - i) / blockSize;
    }
  } else {
    size_t blocksPerQuery = (ref_samples + blockSize - 1) / blockSize;
    blockCount = blocksPerQuery * query_samples;
  }
  return (std::make_tuple(blockSize, blockCount));
}

// Writes a progress meter using the device int which keeps
// track of completed jobs
void reportDistProgress(progress_atomics& progress, long long dist_rows) {
  int now_completed = 0;
  float kern_progress = 0;
  if (dist_rows > progress_blocks) {
    while (now_completed < progress_blocks - 1) {
      if (PyErr_CheckSignals() != 0) {
        progress.set_kill();
        throw py::error_already_set();
      }
      int complete = progress.complete();
      if (complete > now_completed) {
        now_completed = complete;
        kern_progress = now_completed / (float)progress_blocks;
        fprintf(stderr, "%cProgress (GPU): %.1lf%%", 13, kern_progress * 100);
      } else {
        usleep(1000);
      }
    }
  }
}

// Initialise device and return info on its memory
std::tuple<size_t, size_t, size_t> initialise_device(const int device_id) {
  CUDA_CALL(hipSetDevice(device_id));

  size_t mem_free = 0;
  size_t mem_total = 0;
  CUDA_CALL(hipMemGetInfo(&mem_free, &mem_total));
  int shared_size = 0;
  CUDA_CALL(hipDeviceGetAttribute(
      &shared_size, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id));
  return (
      std::make_tuple(mem_free, mem_total, static_cast<size_t>(shared_size)));
}

std::tuple<bool, size_t> check_shared_size(const SketchStrides& strides, const size_t shared_size) {
  size_t sketch_size_bytes =
    strides.sketchsize64 * strides.bbits * sizeof(uint64_t);
  bool use_shared = true;
  if (sketch_size_bytes > shared_size) {
    std::cerr << "You are using a large sketch size, which may slow down "
                "computation on this device"
              << std::endl;
    std::cerr << "Reduce sketch size to "
              << std::floor(64 * shared_size /
                            (strides.bbits * sizeof(uint64_t)))
              << " or less for better performance" << std::endl;
    sketch_size_bytes = 0;
    use_shared = false;
  }
  return std::make_tuple(use_shared, sketch_size_bytes);
}

// Main function to run the distance calculations, reading/writing into
// device_arrays Cache preferences: Upper dist memory access is hard to predict,
// so try and cache as much as possible Query uses on-chip cache (__shared__) to
// store query sketch
std::vector<float> dispatchDists(std::vector<Reference> &ref_sketches,
                                 std::vector<Reference> &query_sketches,
                                 SketchStrides &ref_strides,
                                 SketchStrides &query_strides,
                                 const FlatRandom &flat_random,
                                 const std::vector<uint16_t> &ref_random_idx,
                                 const std::vector<uint16_t> &query_random_idx,
                                 const SketchSlice &sketch_subsample,
                                 const std::vector<size_t> &kmer_lengths,
                                 const bool self, const int cpu_threads,
                                 const size_t shared_size) {
  CUDA_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  // Progress meter
  progress_atomics progress;

  RandomStrides random_strides = std::get<0>(flat_random);
  long long dist_rows;
  if (self) {
    dist_rows =
        (sketch_subsample.ref_size * (sketch_subsample.ref_size - 1)) >> 1;
  } else {
    dist_rows = sketch_subsample.ref_size * sketch_subsample.query_size;
  }

  // Load memory onto device
  DeviceMemory device_arrays(ref_strides, query_strides, ref_sketches,
                             query_sketches, sketch_subsample, flat_random,
                             ref_random_idx, query_random_idx, kmer_lengths,
                             dist_rows, self, cpu_threads);

  bool use_shared;
  size_t shared_size_bytes;
  std::tie(use_shared, shared_size_bytes) =
    check_shared_size(query_strides, shared_size);

  size_t blockSize, blockCount;
  bool max_diagonal = false;
  int dist_col = -1;
  if (self) {
    std::tie(blockSize, blockCount) = getBlockSize(
        sketch_subsample.ref_size, sketch_subsample.ref_size, dist_rows, self);

    // Third argument is the size of __shared__ memory needed by a thread block
    // This is equal to the query sketch size in bytes (at a single k-mer
    // length)
    calculate_dists<<<blockCount, blockSize, shared_size_bytes>>>(
        self, device_arrays.ref_sketches(), sketch_subsample.ref_size,
        device_arrays.ref_sketches(), sketch_subsample.ref_size,
        device_arrays.kmers(), kmer_lengths.size(), device_arrays.dist_mat(),
        dist_rows, device_arrays.random_table(), device_arrays.ref_random(),
        device_arrays.ref_random(), ref_strides, ref_strides, random_strides,
        progress.get_ptrs(), use_shared, dist_col, max_diagonal);
  } else {
    std::tie(blockSize, blockCount) =
        getBlockSize(sketch_subsample.ref_size, sketch_subsample.query_size,
                     dist_rows, self);

    // Third argument is the size of __shared__ memory needed by a thread block
    // This is equal to the query sketch size in bytes (at a single k-mer
    // length)
    calculate_dists<<<blockCount, blockSize, shared_size_bytes>>>(
        self, device_arrays.ref_sketches(), sketch_subsample.ref_size,
        device_arrays.query_sketches(), sketch_subsample.query_size,
        device_arrays.kmers(), kmer_lengths.size(), device_arrays.dist_mat(),
        dist_rows, device_arrays.random_table(), device_arrays.ref_random(),
        device_arrays.query_random(), ref_strides, query_strides,
        random_strides, progress.get_ptrs(), use_shared, dist_col, max_diagonal);
  }

  // Check for error in kernel launch
  CUDA_CALL(hipGetLastError());
  reportDistProgress(progress, dist_rows);
  fprintf(stderr, "%cProgress (GPU): 100.0%%\n", 13);

  // Copy results back to host
  CUDA_CALL(hipDeviceSynchronize());
  std::vector<float> dist_results = device_arrays.read_dists();

  return (dist_results);
}


// Function which sparsifies distances on the fly. Distances are calculated in
// blocks, sorted and top k stored.
// NB cuda graph not needed as API calls faster than ops here
sparse_coo sparseDists(const dist_params params,
  const std::vector<std::vector<uint64_t>> &ref_sketches,
  const std::vector<SketchStrides> &ref_strides,
  const FlatRandom &flat_random,
  const std::vector<uint16_t> &ref_random_idx,
  const std::vector<size_t> &kmer_lengths,
  const int kNN,
  const size_t dist_col,
  const size_t samples_per_chunk,
  const size_t num_big_chunks,
  const int cpu_threads) {
  /*
   *
   *   Device setup
   *
   */
  CUDA_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  // Progress meter (not used for printing in this function)
  progress_atomics progress;

  // const parameters for functions
  const size_t n_chunks = ref_sketches.size();
  const bool self = false;
  const double total_blocks = static_cast<double>(n_chunks) * n_chunks;
  const size_t idx_blockSize = 64;
  const size_t copy_blockSize = 64;
  const int begin_sort_bit = 0;
  const int end_sort_bit = 8 * sizeof(float);

  bool use_shared;
  size_t shared_size_bytes;
  std::tie(use_shared, shared_size_bytes) =
    check_shared_size(ref_strides[0], params.shared_size);

  /*
   *
   *   Memory allocations
   *   All use the big_chunk size, for smaller chunks the end of the alloc
   *   is not used
   *
   */

  // Storage for results on host
  std::vector<float> host_dists(params.n_samples * kNN);
  std::vector<long> i_vec(params.n_samples * kNN);
  std::vector<long> j_vec(params.n_samples * kNN);

  //   sketch block 1
  //   sketch block 2
  //   sketch block 3
  //   sketch block 4
  device_array<uint64_t> block1(ref_sketches[0].size());
  device_array<uint64_t> block2(block1.size());
  device_array<uint64_t> block3(block1.size());
  device_array<uint64_t> block4(block1.size());
  //   dists (chunk size^2)
  //   dists idx (dists.size())
  device_array<float> dists((samples_per_chunk + 1) * (samples_per_chunk + 1));
  device_array<long> dists_idx(dists.size());

  // inner loop sorting
  //   sorted dists (dists.size())
  //   sorted dists idx (dists.size())
  //   offsets
  //   tmp space
  device_array<float> sorted_dists(dists.size());
  device_array<long> sorted_dists_idx(dists.size());
  device_array<int> dist_partitions((samples_per_chunk + 1) + 1);
  device_array<void> dist_sort_tmp;

  // inner loop kNN pick (copy_top_k)
  //   sorted dists
  //   sorted dists idx
  device_array<float> all_sorted_dists(kNN * n_chunks * (samples_per_chunk + 1));
  device_array<long> all_sorted_dists_idx(all_sorted_dists.size());

  // outer loop sorting
  //   doubly sorted dists (staging for final results, copied to host)
  //   doubly sorted dists idx (as above)
  //   offsets
  //   tmp space
  device_array<float> doubly_sorted_dists(all_sorted_dists.size());
  device_array<long> doubly_sorted_dists_idx(all_sorted_dists.size());
  std::vector<int> host_partitions;
  for (int partition_idx = 0; partition_idx < dist_partitions.size(); ++partition_idx) {
    host_partitions.push_back(partition_idx * kNN * n_chunks);
  }
  device_array<int> second_sort_partitions(host_partitions);
  host_partitions.clear();
  device_array<void> outer_dist_sort_tmp;

  // out loop kNN pick (copy_top_k)
  //   final dists (staging for final results, copied to host)
  //   final dists idx (as above)
  device_array<float> final_dists(kNN * (samples_per_chunk + 1));
  device_array<long> final_dists_idx(kNN * (samples_per_chunk + 1));

  //   kmers
  const std::vector<int> kmer_ints(kmer_lengths.begin(), kmer_lengths.end());
  device_array<int> kmers(kmer_ints);

  //   random
  RandomStrides random_strides = std::get<0>(flat_random);
  device_array<float> random_table(std::get<1>(flat_random));
  device_array<uint16_t> random_idx(ref_random_idx);

  /*
   *
   *   Memory setup
   *   Set i_vec on host
   *   Register sketches in host memory
   *   Copy first sketch blocks to device
   *
   */

  // Set i_vec (it just needs to be 0, 0, 0..., 1, 1, 1...)
  #pragma omp parallel for num_threads(cpu_threads)
  for (size_t sample_idx = 0; sample_idx < params.n_samples; ++sample_idx) {
    std::fill_n(i_vec.begin() + sample_idx * kNN, kNN, sample_idx);
  }

  // Register sketches on host so they can be copied async
  for (size_t chunk_idx = 0; chunk_idx < n_chunks; ++chunk_idx) {
    CUDA_CALL(hipHostRegister((void *)ref_sketches[chunk_idx].data(),
                               ref_sketches[chunk_idx].size() * sizeof(uint64_t),
                               hipHostRegisterReadOnly));
  }

  // Four CUDA streams used in loops
  hip_stream dist_stream, idx_stream, mem_stream, sort_stream;

  // Copy first set of refs in. Block 4 is used to store this permanently to stop
  // two copies being needed when entering a new row
  block1.set_array_async(ref_sketches[0].data(), ref_sketches[0].size(), mem_stream.stream());
  block4.set_array_async(block1.data(), block1.size(), mem_stream.stream());

  /*
   *
   *   Loop over chunks
   *   Outer loop over chunks of rows
   *   Inner loop over chunks of columns
   *
   */
  fprintf(stderr, "%cProgress (GPU): %.1lf%%", 13, 0.0f);

  // OUTER LOOP over n_chunks lots of refs
  size_t row_offset = 0;
  for (size_t row_chunk_idx = 0; row_chunk_idx < n_chunks; ++row_chunk_idx) {
    size_t row_samples = samples_per_chunk + (row_chunk_idx < num_big_chunks ? 1 : 0);
    size_t col_offset = 0;

    //  INNER LOOP over n_chunks lots of queries
    for (size_t col_chunk_idx = 0; col_chunk_idx < n_chunks; ++col_chunk_idx) {
      size_t col_samples = samples_per_chunk + (col_chunk_idx < num_big_chunks ? 1 : 0);
      size_t dist_rows = row_samples * col_samples;

      // Check for interrupts
      if (PyErr_CheckSignals() != 0) {
        throw py::error_already_set();
      }

      // Only need to set new sort partitions if moving to a smaller chunk
      if (host_partitions.size() != col_samples + 1) {
        host_partitions.clear();
        for (int partition_idx = 0; partition_idx < col_samples + 1; ++partition_idx) {
          host_partitions.push_back(partition_idx * col_samples);
        }
        dist_partitions.set_array_async(host_partitions.data(), host_partitions.size(), sort_stream.stream());
      }

      //    (stream 1 async) Run dists on 1 vs 2
      size_t blockSize, blockCount;
      std::tie(blockSize, blockCount) =
        getBlockSize(row_samples, col_samples, dist_rows, self);
      uint64_t* query_ptr = col_chunk_idx == 0 ? block4.data() : block2.data();
      bool max_diagonal = col_chunk_idx == row_chunk_idx;
      mem_stream.sync();
      // NB in calculate_dists ref idx changes fastest (so should be the column)
      // so ref and query are 'backwards'
      calculate_dists<<<blockCount, blockSize, shared_size_bytes, dist_stream.stream()>>>(
        self,
        query_ptr, col_samples,
        block1.data(), row_samples,
        kmers.data(), kmers.size(),
        dists.data(), dist_rows,
        random_table.data(), random_idx.data() + col_offset, random_idx.data() + row_offset,
        ref_strides[col_chunk_idx], ref_strides[row_chunk_idx],
        random_strides, progress.get_ptrs(), use_shared, dist_col, max_diagonal
      );

      //    (stream 2 async) Load next into block 3
      //    swap ptrs for block 2 <-> 3
      if (col_chunk_idx + 1 < n_chunks) {
        block3.set_array_async(ref_sketches[col_chunk_idx + 1].data(), ref_sketches[col_chunk_idx + 1].size(), mem_stream.stream());
        block2.swap(block3);
      } else if (row_chunk_idx + 1 < n_chunks) {
        block3.set_array_async(ref_sketches[row_chunk_idx + 1].data(), ref_sketches[row_chunk_idx + 1].size(), mem_stream.stream());
        block1.swap(block3);
      }

      //    (stream 3 async) Set dist idx via kernel
      const size_t idx_blockCount = (dist_rows + idx_blockSize - 1) / idx_blockSize;
      set_idx<<<idx_blockCount, idx_blockSize, 0, idx_stream.stream()>>>(
        dists_idx.data(), row_samples, col_samples, col_offset
      );

      //    (stream 4) hipcub::DeviceSegmentedRadixSort::SortPairs on dists, dists idx -> sorted dists, sorted dists idx
      const int num_items = dist_rows;
      const int num_segments = host_partitions.size() - 1;
      int *d_offsets = dist_partitions.data();
      float *d_keys_in = dists.data();
      float *d_keys_out = sorted_dists.data();
      long *d_values_in = dists_idx.data();
      long *d_values_out = sorted_dists_idx.data();
      // Determine temporary device storage requirements (first run only)
      if (row_chunk_idx == 0 && col_chunk_idx == 0) {
        size_t temp_storage_bytes = 0;
        hipcub::DeviceSegmentedRadixSort::SortPairs(dist_sort_tmp.data(), temp_storage_bytes,
            d_keys_in, d_keys_out, d_values_in, d_values_out,
            num_items, num_segments, d_offsets, d_offsets + 1,
            begin_sort_bit, end_sort_bit, sort_stream.stream());
        dist_sort_tmp.set_size(temp_storage_bytes);
      }
      // Run sorting operation
      dist_stream.sync();
      idx_stream.sync();
      size_t temp_storage_bytes = dist_sort_tmp.size();
      hipcub::DeviceSegmentedRadixSort::SortPairs(dist_sort_tmp.data(), temp_storage_bytes,
          d_keys_in, d_keys_out, d_values_in, d_values_out,
          num_items, num_segments, d_offsets, d_offsets + 1,
          begin_sort_bit, end_sort_bit, sort_stream.stream());

      //    (stream 4) D->D copy of top kNN dists to start of dists
      const bool second_sort = false;
      const size_t dist_out_size = kNN * num_segments;
      const size_t copy_blockCount = (dist_out_size + copy_blockSize - 1) / copy_blockSize;
      copy_top_k<<<copy_blockCount, copy_blockSize, 0, sort_stream.stream()>>>(
        sorted_dists.data(), sorted_dists_idx.data(), all_sorted_dists.data(), all_sorted_dists_idx.data(),
        col_samples, dist_out_size, kNN, col_chunk_idx, n_chunks, second_sort
      );

      // Update progress
      col_offset += col_samples;
      const size_t blocks_done = row_chunk_idx * n_chunks + col_chunk_idx + 1;
      fprintf(stderr, "%cProgress (GPU): %.1lf%%", 13, 100 * blocks_done / total_blocks);
    }

    const int num_items = all_sorted_dists.size();
    const int num_segments = row_samples;
    int *d_offsets = second_sort_partitions.data();
    float *d_keys_in = all_sorted_dists.data();
    float *d_keys_out = doubly_sorted_dists.data();
    long *d_values_in = all_sorted_dists_idx.data();
    long *d_values_out = doubly_sorted_dists_idx.data();
    // Determine temporary device storage requirements
    if (row_chunk_idx == 0) {
      size_t temp_storage_bytes = 0;
      hipcub::DeviceSegmentedRadixSort::SortPairs(outer_dist_sort_tmp.data(), temp_storage_bytes,
          d_keys_in, d_keys_out, d_values_in, d_values_out,
          num_items, num_segments, d_offsets, d_offsets + 1,
          begin_sort_bit, end_sort_bit, sort_stream.stream());
      outer_dist_sort_tmp.set_size(temp_storage_bytes);
    }
    // Run sorting operation
    size_t temp_storage_bytes = outer_dist_sort_tmp.size();
    hipcub::DeviceSegmentedRadixSort::SortPairs(outer_dist_sort_tmp.data(), temp_storage_bytes,
        d_keys_in, d_keys_out, d_values_in, d_values_out,
        num_items, num_segments, d_offsets, d_offsets + 1,
        begin_sort_bit, end_sort_bit, sort_stream.stream());

    // take top kNN
    const bool second_sort = true;
    const int block_offset = 0;
    const size_t dist_out_size = kNN * num_segments;
    const size_t copy_blockCount = (dist_out_size + copy_blockSize - 1) / copy_blockSize;
    copy_top_k<<<copy_blockCount, copy_blockSize, 0, sort_stream.stream()>>>(
      doubly_sorted_dists.data(), doubly_sorted_dists_idx.data(), final_dists.data(), final_dists_idx.data(),
      kNN * n_chunks, dist_out_size, kNN, block_offset, n_chunks, second_sort
    );
    // Copy chunk of results back to host
    final_dists.get_array_async(host_dists.data() + row_offset * kNN, dist_out_size, sort_stream.stream());
    final_dists_idx.get_array_async(j_vec.data() + row_offset * kNN, dist_out_size, sort_stream.stream());

    row_offset += row_samples;
  }
  fprintf(stderr, "%cProgress (GPU): 100.0%%\n", 13);
  CUDA_CALL(hipDeviceSynchronize());

  // Unregister host memory
  for (size_t chunk_idx = 0; chunk_idx < n_chunks; ++chunk_idx) {
    CUDA_CALL(hipHostUnregister((void *)ref_sketches[chunk_idx].data()));
  }

  return (std::make_tuple(i_vec, j_vec, host_dists));
}
